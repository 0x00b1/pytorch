namespace at {
namespace native {
namespace {
const auto elliptic_theta_1_string = jiterator_stringify(
  template<typename T>
  T elliptic_theta_1(T x, T n) {
    return x;
  } // T elliptic_theta_1(T x, T n)
); // elliptic_theta_1_string

const char elliptic_theta_1_name[] = "elliptic_theta_1";

void elliptic_theta_1_cuda_kernel(TensorIteratorBase& iterator) {
#if AT_USE_JITERATOR()
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "elliptic_theta_1_cuda_kernel", [&]() {
    opmath_jitted_gpu_kernel_with_scalars<elliptic_theta_1_name, scalar_t, scalar_t>(iterator, elliptic_theta_1_string);
  });
#else
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "elliptic_theta_1_cuda_kernel", [&]() {
    gpu_kernel_with_scalars(iterator, []GPU_LAMBDA(scalar_t x, scalar_t n) -> scalar_t {
      return special_functions::elliptic_theta_1<scalar_t, true>(x, n);
    });
  });
#endif
} // void elliptic_theta_1_cuda_kernel(TensorIteratorBase& iterator)
} // namespace (anonymous)
REGISTER_DISPATCH(elliptic_theta_1_stub, &elliptic_theta_1_cuda_kernel);
} // namespace native
} // namespace at
