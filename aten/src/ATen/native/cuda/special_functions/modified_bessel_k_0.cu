#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS

#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Math.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/jit_utils.h>
#include <ATen/NumericUtils.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/complex.h>

namespace at {
namespace native {
namespace {
const auto modified_bessel_k_0_string = modified_bessel_i_0_string + jiterator_stringify(
    template<typename T>
    T modified_bessel_k_0(T x) {
        static const T A[] = {
                +1.37446543561352307156e-16,
                +4.25981614279661018399e-14,
                +1.03496952576338420167e-11,
                +1.90451637722020886025e-09,
                +2.53479107902614945675e-07,
                +2.28621210311945178607e-05,
                +1.26461541144692592338e-03,
                +3.59799365153615016266e-02,
                +3.44289899924628486886e-01,
                -5.35327393233902768720e-01,
        };

        static const T B[] = {
                +5.30043377268626276149e-18,
                -1.64758043015242134646e-17,
                +5.21039150503902756861e-17,
                -1.67823109680541210385e-16,
                +5.51205597852431940784e-16,
                -1.84859337734377901440e-15,
                +6.34007647740507060557e-15,
                -2.22751332699166985548e-14,
                +8.03289077536357521100e-14,
                -2.98009692317273043925e-13,
                +1.14034058820847496303e-12,
                -4.51459788337394416547e-12,
                +1.85594911495471785253e-11,
                -7.95748924447710747776e-11,
                +3.57739728140030116597e-10,
                -1.69753450938905987466e-09,
                +8.57403401741422608519e-09,
                -4.66048989768794782956e-08,
                +2.76681363944501510342e-07,
                -1.83175552271911948767e-06,
                +1.39498137188764993662e-05,
                -1.28495495816278026384e-04,
                +1.56988388573005337491e-03,
                -3.14481013119645005427e-02,
                +2.44030308206595545468e+00,
        };

        if (x == T(0.0)) {
            return INFINITY;
        }

        if (x < T(0.0)) {
            return NAN;
        }

        T p;
        T q = 0.0;

        if (x <= T(2.0)) {
            T a = A[0];

            for (uint8_t index = 1; index < 10; index++) {
                p = q;
                q = a;
                a = (x * x - T(2.0)) * q - p + A[index];
            }

            return T(0.5) * (a - p) - log(0.5 * x) * modified_bessel_i_0(x);
        }

        T b = B[0];

        for (uint8_t index = 1; index < 25; index++) {
            p = q;
            q = b;
            b = (T(8.0) / x - T(2.0)) * q - p + B[index];
        }

        return exp(-x) * (T(0.5) * (b - p)) / sqrt(x);
    } // modified_bessel_k_0(T x)
); // modified_bessel_k_0_string

const char modified_bessel_k_0_name[] = "modified_bessel_k_0";

void modified_bessel_k_0_kernel_cuda(TensorIteratorBase &iterator) {
#if AT_USE_JITERATOR()
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "modified_bessel_k_0_cuda", [&]() {
    jitted_gpu_kernel<modified_bessel_k_0_name, scalar_t, scalar_t, 1>(iterator, modified_bessel_k_0_string);
  });
#else
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "modified_bessel_k_0_cuda", [&]() {
    gpu_kernel(iterator, []GPU_LAMBDA(scalar_t x) -> scalar_t {
      return x;
    });
  });
#endif // AT_USE_JITERATOR()
} // void modified_bessel_k_0_kernel_cuda(TensorIteratorBase &iterator)
} // namespace (anonymous)
REGISTER_DISPATCH(special_modified_bessel_k_0_stub, &modified_bessel_k_0_kernel_cuda);
} // namespace native
} // namespace at
