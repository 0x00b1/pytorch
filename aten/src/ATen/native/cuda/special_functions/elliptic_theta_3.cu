namespace at {
namespace native {
namespace {
const auto elliptic_theta_3_string = jiterator_stringify(
  template<typename T>
  T elliptic_theta_3(T x, T n) {
    return x;
  } // T elliptic_theta_3(T x, T n)
); // elliptic_theta_3_string

const char elliptic_theta_3_name[] = "elliptic_theta_3";

void elliptic_theta_3_cuda_kernel(TensorIteratorBase& iterator) {
#if AT_USE_JITERATOR()
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "elliptic_theta_3_cuda_kernel", [&]() {
    opmath_jitted_gpu_kernel_with_scalars<elliptic_theta_3_name, scalar_t, scalar_t>(iterator, elliptic_theta_3_string);
  });
#else
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "elliptic_theta_3_cuda_kernel", [&]() {
    gpu_kernel_with_scalars(iterator, []GPU_LAMBDA(scalar_t x, scalar_t n) -> scalar_t {
      return special_functions::elliptic_theta_3<scalar_t, true>(x, n);
    });
  });
#endif
} // void elliptic_theta_3_cuda_kernel(TensorIteratorBase& iterator)
} // namespace (anonymous)
REGISTER_DISPATCH(elliptic_theta_3_stub, &elliptic_theta_3_cuda_kernel);
} // namespace native
} // namespace at
