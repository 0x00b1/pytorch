namespace at {
namespace native {
namespace {
const auto elliptic_theta_4_string = jiterator_stringify(
  template<typename T>
  T elliptic_theta_4(T x, T n) {
    return x;
  } // T elliptic_theta_4(T x, T n)
); // elliptic_theta_4_string

const char elliptic_theta_4_name[] = "elliptic_theta_4";

void elliptic_theta_4_cuda_kernel(TensorIteratorBase& iterator) {
#if AT_USE_JITERATOR()
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "elliptic_theta_4_cuda_kernel", [&]() {
    opmath_jitted_gpu_kernel_with_scalars<elliptic_theta_4_name, scalar_t, scalar_t>(iterator, elliptic_theta_4_string);
  });
#else
  AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "elliptic_theta_4_cuda_kernel", [&]() {
    gpu_kernel_with_scalars(iterator, []GPU_LAMBDA(scalar_t x, scalar_t n) -> scalar_t {
      return special_functions::elliptic_theta_4<scalar_t, true>(x, n);
    });
  });
#endif
} // void elliptic_theta_4_cuda_kernel(TensorIteratorBase& iterator)
} // namespace (anonymous)
REGISTER_DISPATCH(elliptic_theta_4_stub, &elliptic_theta_4_cuda_kernel);
} // namespace native
} // namespace at
